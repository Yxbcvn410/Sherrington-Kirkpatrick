#include "hip/hip_runtime.h"
/*
 * CudaOperations.cu
 *
 *  Created on: Feb 6, 2019
 *      Author: alexander
 */

#include "Matrice.h"
#include "Spinset.h"
#include "CudaOperator.h"
#include <hip/hip_runtime.h>
#include <sstream>
#include <math.h>

void checkError(hipError_t err, string arg = "") {
	if (err != hipSuccess) {
		cout << "Error: " << hipGetErrorString(err) << endl;
		if (arg != "")
			cout << "Additional data: " << arg << endl;
		std::exit(-1);
	}
}

CudaOperator::CudaOperator(Matrice _matrix, int _blockCount, float _minDiff) {
	minDiff = _minDiff;
	// Set pointers to null
	devSpins = NULL;
	devMat = NULL;
	devUnemptyMat = NULL;
	meanFieldElems = NULL;
	hamiltonianElems = NULL;
	continueIteration = NULL;
	devTemp = NULL;

	size = _matrix.getSize();
	blockSize = 512;
	blockCount = _blockCount;

	hipDeviceProp_t deviceProp;
	checkError(hipGetDeviceProperties(&deviceProp, 0), "getProp");
	blockSize = deviceProp.maxThreadsPerBlock;

	// Allocate memory for pointers at GPU
	checkError(
			hipMalloc((void**) &meanFieldElems,
					sizeof(float) * size * blockCount), "malloc");
	hipMalloc((void**) &devMat, sizeof(float) * size * size);
	hipMalloc((void**) &devSpins, sizeof(float) * size * blockCount);
	hipMalloc((void**) &devUnemptyMat, sizeof(int) * size * (size + 1));
	hipMalloc((void**) &hamiltonianElems, sizeof(double) * size * size);
	hipMalloc((void**) &devTemp, sizeof(float) * blockCount);
	hipMalloc((void**) &continueIteration, sizeof(bool) * _blockCount);

	// Copy model data to GPU memory
	checkError(
			hipMemcpy(devMat, _matrix.getArray(), sizeof(float) * size * size,
					hipMemcpyHostToDevice), "memcpy mat to host");
	hipMemcpy(devUnemptyMat, _matrix.getUnemptyMat(),
			sizeof(int) * size * (size + 1), hipMemcpyHostToDevice);
}

void CudaOperator::cudaLoadSpinset(Spinset spinset, int spinsetIndex) {
	checkError(
			hipMemcpy(&devSpins[spinsetIndex * size], spinset.getArray(),
					sizeof(float) * size, hipMemcpyHostToDevice),
			"memcpy spinset to device");
	hipMemcpy(&devTemp[spinsetIndex], &(spinset.temp), sizeof(float),
			hipMemcpyHostToDevice);
}

void CudaOperator::cudaClear() {
	//Free GPU memory
	hipFree(devSpins);
	hipFree(devMat);
	hipFree(meanFieldElems);
	hipFree(devTemp);
	hipFree(devUnemptyMat);
	hipFree(hamiltonianElems);
	hipFree(continueIteration);
}

__global__ void allocHamiltonian(float* devMat, float* devSpins, int index,
		int size, double* energyTempor) {
	int i;
	int j;

	int wIndex = threadIdx.x + blockIdx.x * blockDim.x;
	while (wIndex < size * size) {
		i = wIndex % size;
		j = (int) (wIndex / size);
		energyTempor[wIndex] = (double) (devSpins[i + index * size]
				* devSpins[j + index * size] * devMat[wIndex]);
		wIndex = wIndex + blockDim.x * gridDim.x;
	}
}

__global__ void quickSum(double* energyTempor, int size) {
	long long offset = 1;
	int wIndex;
	while (offset < size * size) {
		wIndex = threadIdx.x;
		while ((wIndex * 2 + 1) * offset < size * size) {
			energyTempor[wIndex * 2 * offset] += energyTempor[(wIndex * 2 + 1)
					* offset];
			wIndex = wIndex + blockDim.x;
		}
		offset *= 2;
		__syncthreads();
	}
}

double CudaOperator::extractHamiltonian(int index) {
	allocHamiltonian<<<blockCount, blockSize>>>(devMat, devSpins, index, size,
			hamiltonianElems);
	quickSum<<<1, blockSize>>>(hamiltonianElems, size);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	checkError(err, "Kernel at extractEnergy");
	double out;
	checkError(
			hipMemcpy(&out, hamiltonianElems, sizeof(double),
					hipMemcpyDeviceToHost), "memcpy energy to host");
	return out / 2.;
}

Spinset CudaOperator::extractSpinset(int index) {
	float* hSpins = (float*) malloc(sizeof(float) * size);
	checkError(
			hipMemcpy(hSpins, &devSpins[index * size], sizeof(float) * size,
					hipMemcpyDeviceToHost), "memcpy spins to host");
	Spinset outSpins(size);
	for (int i = 0; i < size; i++)
		outSpins.SetSpin(i, hSpins[i]);
	return outSpins;
}

__global__ void cudaKernelPull(float* mat, float* spins, int size, float* temp,
		float tempStep, float* meanFieldElements, bool* continueIteration,
		float minDiff, int* unemptyCells, float linearCoef) {
	int blockId = blockIdx.x;
	int thrId = threadIdx.x;

	do {
		// Lessen temperature
		if (thrId == 0)
			temp[blockId] = temp[blockId] - tempStep;

		// Stabilize
		do {
			__syncthreads();
			// By default current iteration is the last one
			if (thrId == 0)
				continueIteration[blockId] = false;

			for (int spinId = 0; spinId < size; ++spinId) {
				__syncthreads();

				// Transitional value assignment
				int wIndex = thrId;
				while (wIndex < unemptyCells[spinId * (size + 1)]) {
					meanFieldElements[wIndex + blockId * size] =
							spins[unemptyCells[spinId * (size + 1) + wIndex + 1]
									+ blockId * size]
									* mat[spinId * size
											+ unemptyCells[spinId * (size + 1)
													+ wIndex + 1]];
					// BEWARE: Matrix is symmetrical!
					wIndex = wIndex + blockDim.x;
				}
				__syncthreads();

				// Parallelized mean-field computation
				long long offset = 1;
				while (offset < unemptyCells[spinId * (size + 1)]) {
					wIndex = thrId;
					while ((wIndex * 2 + 1) * offset
							< unemptyCells[spinId * (size + 1)]) {
						meanFieldElements[wIndex * 2 * offset + blockId * size] +=
								meanFieldElements[(wIndex * 2 + 1) * offset
										+ blockId * size];
						wIndex = wIndex + blockDim.x;
					}
					offset *= 2;
					__syncthreads();
				}
				__syncthreads();

				// Mean-field calculation complete - write new spin and delta
				if (thrId == 0) {
					float meanField = meanFieldElements[blockId * size];
					float old = spins[spinId + blockId * size];
					if (temp[blockId] > 0) {
						spins[spinId + blockId * size] = -1
								* tanh(meanField / temp[blockId]) * linearCoef
								+ spins[spinId + blockId * size]
										* (1 - linearCoef);
					} else if (meanField > 0)
						spins[spinId + blockId * size] = -1;
					else
						spins[spinId + blockId * size] = 1;

					if (minDiff < fabs(old - spins[spinId + blockId * size]))
						continueIteration[blockId] = true; // Too big delta. One more iteration needed
				}
				__syncthreads();
			}
		} while (continueIteration[blockId]);
	} while (temp[blockId] >= 0);
}

void CudaOperator::cudaPull(float pStep, float linearCoef) {
	cudaKernelPull<<<blockCount, blockSize>>>(devMat, devSpins, size, devTemp,
			pStep, meanFieldElems, continueIteration, minDiff, devUnemptyMat, linearCoef);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	checkError(err, "Kernel at cudaPull");
}
