#include "hip/hip_runtime.h"
#include "Matrix.h"
#include "Spinset.h"
#include "CudaOperator.h"
#include <hip/hip_runtime.h>
#include <sstream>
#include <math.h>

void checkError(hipError_t err, string arg = "") {
	if (err != hipSuccess) {
		cout << "Error: " << hipGetErrorString(err) << endl;
		if (arg != "")
			cout << "Additional data: " << arg << endl;
		std::exit(-1);
	}
}

CudaOperator::CudaOperator(Matrix matrix, int blockCnt, float _minDiff) {
	minDiff = _minDiff;
	// Set pointers to null
	devSpins = NULL;
	devMat = NULL;
	devUnemptyMat = NULL;
	meanFieldElems = NULL;
	energyElems = NULL;
	continueIteration = NULL;
	devTemp = NULL;

	size = matrix.getSize();
	blockSize = 512;
	blockCount = blockCnt;

	hipDeviceProp_t deviceProp;
	checkError(hipGetDeviceProperties(&deviceProp, 0), "getProp");
	blockSize = deviceProp.maxThreadsPerBlock;

	// Allocate memory for pointers at GPU
	checkError(hipMalloc((void**) &meanFieldElems, sizeof(float) * size * blockCount), "malloc");
	hipMalloc((void**) &devMat, sizeof(float) * size * size);
	hipMalloc((void**) &devSpins, sizeof(float) * size * blockCount);
	hipMalloc((void**) &devUnemptyMat, sizeof(int) * size * (size + 1));
	hipMalloc((void**) &energyElems, sizeof(double) * size * size);
	hipMalloc((void**) &devTemp, sizeof(float) * blockCount);
	hipMalloc((void**) &continueIteration, sizeof(bool) * blockCnt);

	// Copy model data to GPU memory
	checkError(hipMemcpy(devMat, matrix.getArray(), sizeof(float) * size * size, hipMemcpyHostToDevice),
			"memcpy mat to host");
	hipMemcpy(devUnemptyMat, matrix.getUnemptyMat(), sizeof(int) * size * (size + 1), hipMemcpyHostToDevice);
}

void CudaOperator::cudaLoadSpinset(Spinset spinset, int index) {
	checkError(hipMemcpy(&devSpins[index * size], spinset.getArray(), sizeof(float) * size, hipMemcpyHostToDevice),
			"memcpy spinset to device");
	hipMemcpy(&devTemp[index], &(spinset.temp), sizeof(float), hipMemcpyHostToDevice);
}

void CudaOperator::cudaClear() {
	//Free GPU memory
	hipFree(devSpins);
	hipFree(devMat);
	hipFree(meanFieldElems);
	hipFree(devTemp);
	hipFree(devUnemptyMat);
	hipFree(energyElems);
	hipFree(continueIteration);
}

__global__ void allocHamiltonian(float* devMat, float* devSpins, int index, int size, double* energyTempor) {
	int i;
	int j;

	int wIndex = threadIdx.x;
	while (wIndex < size * size) {
		i = wIndex % size;
		j = (int) (wIndex / size);
		energyTempor[wIndex] = (double) (devSpins[i + index * size] * devSpins[j + index * size] * devMat[wIndex]);
		wIndex = wIndex + blockDim.x;
	}
}

__global__ void quickSum(double* energyTempor, int size) {
	long long offset = 1;
	int wIndex;
	while (offset < size * size) {
		wIndex = threadIdx.x;
		while ((wIndex * 2 + 1) * offset < size * size) {
			energyTempor[wIndex * 2 * offset] += energyTempor[(wIndex * 2 + 1) * offset];
			wIndex = wIndex + blockDim.x;
		}
		offset *= 2;
		__syncthreads();
	}
}

double CudaOperator::extractHamiltonian(int index) {
	allocHamiltonian<<<1, blockSize>>>(devMat, devSpins, index, size,
			energyElems);
	quickSum<<<1, blockSize>>>(energyElems, size);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	checkError(err, "Kernel at extractEnergy");
	double out;
	checkError(
			hipMemcpy(&out, energyElems, sizeof(double),
					hipMemcpyDeviceToHost), "memcpy energy to host");
	return out;
}

Spinset CudaOperator::extractSpinset(int index) {
	float* hSpins = (float*) malloc(sizeof(float) * size);
	checkError(hipMemcpy(hSpins, &devSpins[index * size], sizeof(float) * size, hipMemcpyDeviceToHost),
			"memcpy spins to host");
	Spinset outSpins(size);
	for (int i = 0; i < size; i++)
		outSpins.SetSpin(i, hSpins[i]);
	return outSpins;
}

__global__ void cudaKernelPull(float* mat, float* spins, int size, float* temp, float tempStep,
		float* meanFieldElements, bool* continueIteration, float minDiff, int* unemptyCells) {
	int blockId = blockIdx.x;
	int thrId = threadIdx.x;

	do {
		// Lessen temperature
		if (thrId == 0)
			temp[blockId] = temp[blockId] - tempStep;

		// Stabilize
		do {
			__syncthreads();
			// By default current iteration is the last one
			if (thrId == 0)
				continueIteration[blockId] = false;

			for (int spinId = 0; spinId < size; ++spinId) {
				__syncthreads();

				// Transitional value assignment
				int wIndex = thrId;
				while (wIndex < unemptyCells[spinId * (size + 1)]) {
					meanFieldElements[unemptyCells[spinId * (size + 1) + wIndex + 1] + blockId * size] =
							spins[unemptyCells[spinId * (size + 1) + wIndex + 1] + blockId * size]
									* mat[spinId * size + unemptyCells[spinId * (size + 1) + wIndex + 1]];
					// BEWARE: Matrix is symmetrical!
					wIndex = wIndex + blockDim.x;
				}
				__syncthreads();

				// Parallelized mean-field computation
				float meanField = 0;
				long long offset = 1;
				while (offset < unemptyCells[spinId * (size + 1)]) {
					wIndex = thrId;
					while ((wIndex * 2 + 1) * offset < unemptyCells[spinId * (size + 1)]) {
						meanFieldElements[unemptyCells[spinId * (size + 1) + wIndex * 2 * offset + 1] + blockId * size] +=
								meanFieldElements[unemptyCells[spinId * (size + 1) + (wIndex * 2 + 1) * offset + 1]
										+ blockId * size];
						wIndex = wIndex + blockDim.x;
					}
					offset *= 2;
					__syncthreads();
				}
				if (thrId == 0)
					meanField = meanFieldElements[blockId * size + unemptyCells[spinId * (size + 1) + 1]];

				// Mean-field calculation complete - write new spin and delta
				if (thrId == 0) {
					float old = spins[spinId + blockId * size];
					if (temp[blockId] > 0) {
						spins[spinId + blockId * size] = -1 * tanh(meanField / temp[blockId]);
					} else if (meanField > 0)
						spins[spinId + blockId * size] = -1;
					else
						spins[spinId + blockId * size] = 1;

					if (minDiff < fabs(old - spins[spinId + blockId * size]))
						continueIteration[blockId] = true; // Too big delta. One more iteration needed
				}
				__syncthreads();
			}
		} while (continueIteration[blockId]);
	} while (temp[blockId] >= 0);
}

void CudaOperator::cudaPull(float pStep) {
	cudaKernelPull<<<blockCount, blockSize>>>(devMat, devSpins, size, devTemp,
			pStep, meanFieldElems, continueIteration, minDiff, devUnemptyMat);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	checkError(err, "Kernel at cudaPull");
}
