#include "hip/hip_runtime.h"
#include "Matrix.h"
#include "Spinset.h"
#include "CudaOperations.h"
#include <hip/hip_runtime.h>

double CudaOperations::getEnergy(Matrix matrix, double* spinset) {
}

__global__ void calcF(double *matrix, double *spinset, int* spinIndex,
		int* size, double *calc) {
	int i = threadIdx.x;
	if (i >= *size || i == *spinIndex)
		return;
	if (i > *spinIndex)
		calc[i] = spinset[i] * matrix[*spinIndex * *size + i];
	else
		calc[i] = spinset[i] * matrix[i * *size + *spinIndex];
}

void checkError(hipError_t err, string arg = ""){
	if(err != hipSuccess){
		cout << "Error: "<< hipGetErrorString(err)<<endl;
		if(arg != "")
			cout << "Additional data: " << arg << endl;
		std::exit(-1);
	}
}

double CudaOperations::getForce(Matrix matrix, double* spinset, int spinIndex) {
	//Define
	double* outp = NULL;
	double* dMat = NULL;
	double* dSpins = NULL;
	int* dSInd = NULL;
	int* size = NULL;

	//Allocate memory in GPU
	checkError(hipMalloc((void **) &outp, sizeof(double) * matrix.getSize()), "malloc");
	hipMalloc((void**) &dMat, sizeof(double) * matrix.getSize());
	hipMalloc((void**) &dSpins,
			sizeof(double) * matrix.getSize() * matrix.getSize());
	hipMalloc((void**) &dSInd, sizeof(int));
	hipMalloc((void**) &size, sizeof(int));

	//Copy data
	checkError(hipMemcpy(dMat, matrix.getArray(), sizeof(double) * matrix.getSize(),
			hipMemcpyHostToDevice), "memcpy");
	hipMemcpy(dSpins, spinset,
			sizeof(double) * matrix.getSize() * matrix.getSize(),
			hipMemcpyHostToDevice);
	hipMemcpy(dSInd, &spinIndex, sizeof(int), hipMemcpyHostToDevice);
	int* ss = (int*) malloc(sizeof(int));
	ss[0] = matrix.getSize();
	hipMemcpy(size, ss, sizeof(int), hipMemcpyHostToDevice);

	//Start
	calcF<<<1, matrix.getSize()>>>(dMat, dSpins, dSInd, size, outp);

	hipDeviceSynchronize();
	double result = 0;
	double* ress = new double[matrix.getSize()];
	hipMemcpy(ress, outp, sizeof(double) * matrix.getSize(),
			hipMemcpyDeviceToHost);
	for (int i = 0; i < matrix.getSize(); ++i) {
		result += ress[i];
	}

	//Free memory of GPU
	hipFree(outp);
	hipFree(dMat);
	hipFree(dSpins);
	hipFree(dSInd);
	hipFree(size);

	//Free CPU memory too
	free(ress);

	return result;
}
