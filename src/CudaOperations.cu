#include "hip/hip_runtime.h"
#include "Matrix.h"
#include "Spinset.h"
#include "CudaOperations.h"
#include <hip/hip_runtime.h>
#include <math.h>

//GPU memory pointers
double* devSpins = NULL; //Spinset
double* devMat = NULL; //Matrix
double* forceElems = NULL; //Temporary storage for counting force
int* devSize = NULL; //Size
double* devTemp = NULL; //Temperature
double* diff = NULL;
double* energyMat1 = NULL;
double* energyMat2 = NULL;
double* energy = NULL;

//CPU variables
double temp;
int size;

void checkError(hipError_t err, string arg = "") {
	if (err != hipSuccess) {
		cout << "Error: " << hipGetErrorString(err) << endl;
		if (arg != "")
			cout << "Additional data: " << arg << endl;
		std::exit(-1);
	}
}

void CudaOperations::cudaInit(Matrix matrix) {
	// Set pointers to null
	devSpins = NULL;
	devMat = NULL;
	forceElems = NULL;
	devSize = NULL;
	energyMat1 = NULL;
	energyMat2 = NULL;
	diff = NULL;
	devTemp = NULL;

	size = matrix.getSize();

	// Allocate memory for pointers at GPU
	checkError(hipMalloc((void**) &forceElems, sizeof(double) * size),
			"malloc");
	hipMalloc((void**) &devMat, sizeof(double) * size * size);
	hipMalloc((void**) &devSpins, sizeof(double) * size);
	hipMalloc((void**) &devSize, sizeof(int));
	hipMalloc((void**) &energyMat1, sizeof(double) * size * size);
	hipMalloc((void**) &energyMat2, sizeof(double) * size);
	hipMalloc((void**) &energy, sizeof(double));
	hipMalloc((void**) &devTemp, sizeof(double));
	hipMalloc((void**) &diff, sizeof(double));

	// Copy model data to GPU memory
	hipMemcpy(devMat, matrix.getArray(), sizeof(double) * size * size,
			hipMemcpyHostToDevice);
	hipMemcpy(devSize, &size, sizeof(int), hipMemcpyHostToDevice);
}

void CudaOperations::cudaLoadSpinset(Spinset spinset) {
	checkError(
			hipMemcpy(devSpins, spinset.getArray(), sizeof(double) * size,
					hipMemcpyHostToDevice));
	hipMemcpy(devTemp, &(spinset.temp), sizeof(double),
			hipMemcpyHostToDevice);
	temp = spinset.temp;
	cout << "spinset loaded" << endl << spinset.getSpins() << endl << temp
			<< endl;
}

void CudaOperations::cudaClear() {
	//Free GPU memory
	hipFree(devSpins);
	hipFree(devMat);
	hipFree(forceElems);
	hipFree(devSize);
	hipFree(devTemp);
	hipFree(energy);
	hipFree(energyMat1);
	hipFree(energyMat2);
	hipFree(diff);
}

__global__ void calcEnergy1(double* devMat, double* devSpins, int* devSize,
		double* energyMat1) {
	int i = threadIdx.x, j = threadIdx.y;
	energyMat1[i * *devSize + j] = devSpins[i] * devSpins[j]
			* devMat[i * *devSize + j];
}

__global__ void calcEnergy2(double* energyMat1, double* energyMat2,
		int* devSize) {
	int inn = threadIdx.x;
	energyMat2[inn] = 0;
	for (int i = 0; i < *devSize; ++i) {
		energyMat2[inn] += energyMat1[inn * *devSize + i];
	}
}

__global__ void calcEnergy3(double* energyMat2, double* energy, int* devSize) {
	*energy = 0;
	for (int i = 0; i < *devSize; ++i) {
		*energy += energyMat2[i];
	}
}

double CudaOperations::extractEnergy() {
	calcEnergy1<<<1, dim3(size, size)>>>(devMat, devSpins, devSize, energyMat1);
	calcEnergy2<<<1, size>>>(energyMat1, energyMat2, devSize);
	calcEnergy3<<<1, 1>>>(energyMat2, energy, devSize);
	hipDeviceSynchronize();
	double out;
	checkError(hipMemcpy(&out, energy, sizeof(double), hipMemcpyDeviceToHost),
			"energy memcpy");
	return out;
}

__global__ void cudaStabilize(double* mat, double* spins, int* size,
		double* temp, double* forceElements, double* diff, int* itC) {
	//Launch in size threads
	int thrId = threadIdx.x;
	if (thrId >= *size) {
		*itC = -1;
		return;
	}
	if (thrId == 1)
	*itC = 0;

	while (true) {
		//Iterate on all spins
		if (thrId == 0) {
			*diff = 0;
			*itC++;
		}
		for (int spinId = 0; spinId < *size; ++spinId) {
			forceElements[thrId] = mat[spinId * *size + thrId];
			__syncthreads();

			//Here you will be able to see the hellish code for calculating the sum of an array in log(N) time
			if (thrId == 0) {
				// Calculate force...
				double force = 0;
				for (int i = 0; i < *size; ++i) {
					force += forceElements[i];
				}

				// Calculate new spin...
				double old = spins[spinId];
				if (*temp > 0)
					spins[spinId] = -1 * tanh(force / *temp);
				else if (force > 0)
					spins[spinId] = -1;
				else if (force < 0)
					spins[spinId] = 1;
				else
					spins[spinId] = 0;

				//And refresh diff
				if (*diff < abs(old - spins[spinId]))
					*diff = abs(old - spins[spinId]);
			}
		}

		__syncthreads();
		if (*diff < 0.000001)
			return; // diff link is same for all threads; Terminate all if diff is appropriate
	}
}

void CudaOperations::cudaPull(double pStep) {
	hipMemcpy(devTemp, &temp, sizeof(double), hipMemcpyHostToDevice);
	int* itC;
	hipMalloc((void**) &itC, sizeof(int));
	cudaStabilize<<<1, size>>>(devMat, devSpins, devSize, devTemp, forceElems,
			diff, itC);
	hipDeviceSynchronize();
	int ill;
	hipMemcpy(&ill, itC, sizeof(int), hipMemcpyDeviceToHost);
	cout << ill;
	do {
		temp -= pStep;
		checkError(
				hipMemcpy(devTemp, &temp, sizeof(double),
						hipMemcpyHostToDevice), "memcpy temperature");
		cudaStabilize<<<1, size>>>(devMat, devSpins, devSize, devTemp,
				forceElems, diff, itC);
		hipDeviceSynchronize();
		int ill;
		hipMemcpy(&ill, itC, sizeof(int), hipMemcpyDeviceToHost);
		cout << ill;
	} while (temp > 0);
	double* spinset = new double[size];
	hipMemcpy(spinset, devSpins, sizeof(double) * size,
			hipMemcpyDeviceToHost);
	cout << "stable:" << endl;
	for (int i = 0; i < size; i++)
		cout << spinset[i] << " ";
	cout << endl;
}
