#include "hip/hip_runtime.h"
#include "Matrix.h"
#include "Spinset.h"
#include "CudaOperations.h"

double CudaOperations::getEnergy(Matrix matrix, double* spinset) {
}

__global__ void calcF(double *matrix, double *spinset, int spinIndex, int size,
		double *calc) {
	int i = threadIdx.x;
	if (i >= size || i == spinIndex)
		return;
	if (i > spinIndex)
		calc[spinIndex] = spinset[spinIndex] * matrix[spinIndex * size + i];
	else
		calc[spinIndex] = spinset[spinIndex] * matrix[i * size + spinIndex];
}

double CudaOperations::getForce(Matrix matrix, double* spinset, int spinIndex) {
	//Define
	double* outp = NULL;
	double* dMat = NULL;
	double* dSpins = NULL;
	int dSInd = NULL;
	int size = NULL;

	//Allocate memory in GPU
	hipError_t r = hipMalloc((void **)&outp, sizeof(double) * matrix.getSize());
	cout << r;
	hipMalloc(&dMat, sizeof(double) * matrix.getSize());
	hipMalloc(&dSpins, sizeof(double) * matrix.getSize() * matrix.getSize());
	hipMalloc((void**) &dSInd, sizeof(int));
	hipMalloc((void**) &size, sizeof(int));

	//Copy data
	hipMemcpy(dMat, matrix.getArray(), sizeof(dMat), hipMemcpyHostToDevice);
	hipMemcpy(dSpins, spinset, sizeof(dSpins), hipMemcpyHostToDevice);
	hipMemcpy(&dSInd, &spinIndex, sizeof(dSInd), hipMemcpyHostToDevice);
	int ss = matrix.getSize();
	hipMemcpy(&size, &ss, sizeof(size), hipMemcpyHostToDevice);

	//Start
	calcF<<<1, matrix.getSize()>>>(dMat, dSpins, dSInd, size, outp);

	hipDeviceSynchronize();
	double result = 0;
	double* ress = new double[matrix.getSize()];
	hipMemcpy(ress, outp, sizeof(double) * matrix.getSize(),
			hipMemcpyDeviceToHost);
	for (int i = 0; i < matrix.getSize(); i++) {
		result += ress[i];
	}

	//Free memory of GPU
	hipFree(outp);
	hipFree(dMat);
	hipFree(dSpins);
	hipFree(&dSInd);
	hipFree(&size);
	return result;
}
