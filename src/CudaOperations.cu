#include "hip/hip_runtime.h"
#include "Matrix.h"
#include "Spinset.h"
#include "CudaOperations.h"
#include <hip/hip_runtime.h>

double CudaOperations::getEnergy(Matrix matrix, double* spinset) {
}

__global__ void calcF(double *matrix, double *spinset, int* spinIndex,
		int* size, double *calc) {
	int i = threadIdx.x;
	if (i >= size[0])
		return;
	if (i > spinIndex[0])
		calc[i] = spinset[i] * matrix[(spinIndex[0]) * (size[0]) + i];
	else
		calc[i] = spinset[i] * matrix[i * (size[0]) + spinIndex[0]];
}

void checkError(hipError_t err, string arg = "") {
	if (err != hipSuccess) {
		cout << "Error: " << hipGetErrorString(err) << endl;
		if (arg != "")
			cout << "Additional data: " << arg << endl;
		std::exit(-1);
	}
}

double CudaOperations::getForce(Matrix matrix, double* spinset, int spinIndex) {
	//Define
	double* outp = NULL;
	double* dMat = NULL;
	double* dSpins = NULL;
	int* dSInd = NULL;
	int* size = NULL;

	//Allocate memory in GPU
	checkError(hipMalloc((void **) &outp, sizeof(double) * matrix.getSize()),
			"malloc");
	hipMalloc((void**) &dMat, sizeof(double) * matrix.getSize() * matrix.getSize());
	hipMalloc((void**) &dSpins,
			sizeof(double) * matrix.getSize());
	hipMalloc((void**) &dSInd, sizeof(int));
	hipMalloc((void**) &size, sizeof(int));

	//Copy data
	checkError(
			hipMemcpy(dMat, matrix.getArray(),
					sizeof(double) * matrix.getSize() * matrix.getSize(), hipMemcpyHostToDevice),
			"memcpy");
	hipMemcpy(dSpins, spinset,
			sizeof(double) * matrix.getSize(),
			hipMemcpyHostToDevice);
	hipMemcpy(dSInd, &(spinIndex), sizeof(int), hipMemcpyHostToDevice);
	int* ss = (int*) malloc(sizeof(int));
	*ss = matrix.getSize();
	hipMemcpy(size, ss, sizeof(int), hipMemcpyHostToDevice);

	//Start
	calcF<<<1, *ss>>>(dMat, dSpins, dSInd, size, outp);

	hipDeviceSynchronize();
	double result = 0;
	double* ress = new double[*ss];
	checkError(
			hipMemcpy(ress, outp, sizeof(double) * (*ss),
					hipMemcpyDeviceToHost), "memcpy back");
	for (int i = 0; i < matrix.getSize(); ++i) {
		result += ress[i];
	}
	//Free memory of GPU
	hipFree(outp);
	hipFree(dMat);
	hipFree(dSpins);
	hipFree(dSInd);
	hipFree(size);

	//Free CPU memory too
	free(ress);

	return result;
}
