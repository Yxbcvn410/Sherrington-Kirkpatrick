#include "hip/hip_runtime.h"
#include "Matrix.h"
#include "Spinset.h"
#include "CudaOperations.h"
#include <hip/hip_runtime.h>
#include <cmath>

//GPU memory pointers
double* devSpins = NULL;
double* devMat = NULL;
double* forceElems = NULL;
int* devSize = NULL;
int* devSpinIndex = NULL;

//CPU variables
double* hForceElems;
double temp;
int size;

__device__ volatile int sem = 0;

__device__ void acquire_semaphore(volatile int *lock) {
	while (atomicCAS((int *) lock, 0, 1) != 0)
		;
}

__device__ void release_semaphore(volatile int *lock) {
	*lock = 0;
	__threadfence();
}

void checkError(hipError_t err, string arg = "") {
	if (err != hipSuccess) {
		cout << "Error: " << hipGetErrorString(err) << endl;
		if (arg != "")
			cout << "Additional data: " << arg << endl;
		std::exit(-1);
	}
}

void CudaOperations::cudaInit(Matrix matrix) {
	// Set pointers to null
	devSpins = NULL;
	devMat = NULL;
	forceElems = NULL;
	devSize = NULL;
	devSpinIndex = NULL;

	size = matrix.getSize();

	// Allocate memory for pointers at GPU
	checkError(hipMalloc((void**) &forceElems, sizeof(double) * modelSize),
			"malloc");
	hipMalloc((void**) &oForce, sizeof(double));
	hipMalloc((void**) &devMat, sizeof(double) * modelSize * modelSize);
	hipMalloc((void**) &devSpins, sizeof(double) * modelSize);
	hipMalloc((void**) &devSize, sizeof(int));
	hipMalloc((void**) &devSpinIndex, sizeof(int));

	//Allocate memory for CPU pointers
	hForceElems = (double*) malloc(sizeof(double) * size);

	// Copy model data to GPU memory
	hipMemcpy(devMat, matrix.getArray(), sizeof(double) * size * size,
			hipMemcpyHostToDevice);
	hipMemcpy(devSize, &size, sizeof(int), hipMemcpyHostToDevice);
}

void CudaOperations::cudaLoadSpinset(Spinset spinset) {
	checkError(
			hipMemcpy(devSpins, spinset.getArray(), sizeof(double) * size,
					hipMemcpyHostToDevice));
	temp = spinset.temp;
}

void CudaOperations::cudaClear() {
	//Free GPU memory
	hipFree(devSpins);
	hipFree(devMat);
	hipFree(forceElems);
	hipFree (oForces);
	hipFree(devSize);
}

double extractPreferredVal() {
	checkError(
			hipMemcpy(hForceElems, forceElems, sizeof(double) * size,
					hipMemcpyDeviceToHost), "Memcpy force elements");
	double force = 0;
	for (int i = 0; i < size; i++)
		force += hForceElems[i];
	if (temp <= 0) {
		if (getForce(index, matrix) > 0)
			return -1;
		else if (getForce(index, matrix) < 0)
			return 1;
		else
			return 0;
	} else
		return tanh((-1 * getForce(index, matrix)) / temp);
}

__global__ void cuGetForce() {
	int i = threadIdx.x;
	if (i == *devSpinIndex || i >= devSize)
		return;
	if (i < *devSpinIndex)
		forceElems[i] = devSpins[i] * devMat[i * *devSize + *devSpinIndex];
	else
		forceElems[i] = devSpins[i] * devMat[*devSpinIndex * *devSize + i];
}

double CudaOperations::cudaIterate() {
	double diff = 0;
	int spinIndex = 0;
	for (spinIndex = 0; spinIndex < size; ++spinIndex) {
		hipMemcpy(devSpinIndex, &spinIndex, sizeof(int),
				hipMemcpyHostToDevice);

	}
}
